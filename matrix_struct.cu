#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "matrix_struct.h"

Matrix* makeMatrix(int row, int col, int device_type){
    Matrix *mat = (Matrix*)malloc(sizeof(Matrix)); 
    if(!device_type){
        mat->M = (float*)malloc(row * col * sizeof(float));
    }else{
        hipSetDevice(device_type-1);
        hipMalloc(&mat->M, row * col * sizeof(float));    
    }
    
    mat->device_type = device_type;
    mat->row = row;
    mat->col = col;
    return mat;
}

void freeMatrix(Matrix *mat) {
    if (mat == NULL) {
        fprintf(stderr, "Attempted to free a NULL matrix pointer\n");
        return;
    }
    if (mat->M == NULL) {
        fprintf(stderr, "Matrix data pointer is NULL\n");
    } else {
        if (mat->device_type) {
            hipSetDevice(mat->device_type - 1);
            hipError_t err = hipFree(mat->M);
            if (err != hipSuccess) {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
            }
        } else {
            free(mat->M);
        }
    }
    free(mat);
}


void printMatrix(Matrix *mat){
    if(mat->device_type){
        printf("GPU mem can not be printed");
        return;
    }
    for(int i=0; i < mat->row; i++){
        for(int j=0; j < mat->col; j++){
            printf("%f\t", mat->M[i * mat->col + j]);
        }
        printf("\n");
    }
}

void infoMatrix(Matrix *mat){
    printf("device : %d dim:(%d, %d)\n", mat->device_type,mat->row, mat->col);
}

Matrix* copyMatToDevice(Matrix *mat, int device_type){
    if(!mat->device_type){
        hipSetDevice(device_type-1);
        Matrix *dMat = makeMatrix(mat->row, mat->col, device_type);
        hipMemcpy(dMat->M, mat->M, mat->row * mat->col * sizeof(float), hipMemcpyHostToDevice);
        return dMat;
    }
    return NULL;
}

Matrix* copyMatToHost(Matrix *dMat){
    if(dMat->device_type){
        Matrix * mat = makeMatrix(dMat->row, dMat->col, 0);
        hipSetDevice(dMat->device_type-1);
        hipMemcpy(mat->M, dMat->M, mat->row * mat->col * sizeof(float), hipMemcpyDeviceToHost);
        return mat;
    }
    return NULL;
}

Matrix* copyMatrix(Matrix *mat, int device_type){
    if(!device_type){//device_type == cpu
            Matrix * mat_copy = makeMatrix(mat->row, mat->col, 0);
            hipSetDevice(mat->device_type-1);
            hipMemcpy(mat_copy->M, mat->M, mat_copy->row * mat_copy->col * sizeof(float), hipMemcpyDeviceToHost);
            return mat_copy;
    }else {
        if(!mat->device_type){
            int tmp;
            hipGetDeviceCount(&tmp);
            if(device_type > 0 && device_type <= tmp){
                return copyMatToDevice(mat, device_type);
            }else{
                printf("invalid device type\n");
                return NULL;
            }
        }else{
            //여기에 devicetodevice를 조진다.
            printf("invalid device type\n");
            return NULL;
        }
    }
}
Matrix* moveMatrix(Matrix *mat, int device_type){
    Matrix *tmp_mat = copyMatrix(mat, device_type);
    if(!tmp_mat)
        return NULL;
    freeMatrix(mat);
    return tmp_mat;
}

Matrix* copyMatrix_inline(Matrix *dst, Matrix* src, int device_type){
    if(dst->col == src->col && dst->row == src->row){
        printf("two Matrices has different row or column.\n");
        return NULL;
    }
    if(!devicetype){
        hipSetDevice(src-> device_type-1);
        hipMemcpy(dst->M, src->M, sizeof(float) * row * col, hipMemcpyDeviceToHost);
    }else{
        if(device_type > 0 && device_type <= tmp){
            return copyMatToDevice(mat, device_type);
        }else{
            printf("invalid device type\n");
            return NULL;
        }
    }
}


__global__ void tiledMM(float *A, float *B, float *C, float *bias, int M, int N, int K) {
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float s_a[tile_SIZE][tile_SIZE];
    __shared__ float s_b[tile_SIZE][tile_SIZE];

    float tmp = 0.0f;

    for (int i = 0; i < (K + tile_SIZE - 1) / tile_SIZE; i++) {
        if (row < M && (i * tile_SIZE + threadIdx.x) < K)
            s_a[threadIdx.y][threadIdx.x] = A[row * K + (i * tile_SIZE + threadIdx.x)];
        else
            s_a[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && (i * tile_SIZE + threadIdx.y) < K)
            s_b[threadIdx.y][threadIdx.x] = B[(i * tile_SIZE + threadIdx.y) * N + col];
        else
            s_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int j = 0; j < tile_SIZE; j++) {
            tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        if (bias)
            C[row * N + col] = tmp + bias[col];
        else
            C[row * N + col] = tmp;
    }
}

void matmul_(float *dA, float *dB, float *dC, float *dBias, int M, int N, int K) {
    dim3 dimGrid((N + tile_SIZE - 1) / tile_SIZE, (M + tile_SIZE - 1) / tile_SIZE);
    dim3 dimBlock(tile_SIZE, tile_SIZE);
    tiledMM<<<dimGrid, dimBlock>>>(dA, dB, dC, dBias, M, N, K);
}

Matrix *matmul_Bias(Matrix *dA, Matrix *dB, Matrix *dBias) {
    if (dA->device_type != dB->device_type || dA->device_type != dBias->device_type) {
        printf("two Matrix is on different device. dA : %d, dB: %d\n", dA->device_type, dB->device_type);
        return NULL;
    }
    if (dBias->col != dB->col) {
        printf("dBias and dB should have same num of columns\n");
        return NULL;
    }
    if (dBias->row != 1) {
        printf("dBias should have only one row.\n");
        return NULL;
    }
    if (dA->col != dB->row) {
        printf("number of column of dA and row of dB doesn't match\n");
        return NULL;
    }
    Matrix *dC = makeMatrix(dA->row, dB->col, dA->device_type);
    hipSetDevice(dA->device_type - 1);
    matmul_(dA->M, dB->M, dC->M, dBias->M, dA->row, dB->col, dA->col);
    return dC;
}

Matrix *matmul_Bias_inline(Matrix *dC, Matrix *dA, Matrix *dB, Matrix *dBias) {
    if (dA->device_type != dB->device_type || dA->device_type != dBias->device_type) {
        printf("two Matrix is on different device. dA : %d, dB: %d\n", dA->device_type, dB->device_type);
        return NULL;
    }
    if (dBias->col != dB->col) {
        printf("dBias and dB should have same num of columns\n");
        return NULL;
    }
    if (dBias->row != 1) {
        printf("dBias should have only one row.\n");
        return NULL;
    }
    if (dA->col != dB->row) {
        printf("number of column of dA and row of dB doesn't match\n");
        return NULL;
    }
    if (dC->device_type != dA->device_type) {
        printf("result matrix is on different device.\n");
        return NULL;
    }
    if (dC->row != dA->row || dC->col != dB->col) {
        printf("result matrix is in different dimension.\n");
        return NULL;
    }
    hipSetDevice(dA->device_type - 1);
    matmul_(dA->M, dB->M, dC->M, dBias->M, dA->row, dB->col, dA->col);
    return dC;
}

Matrix *matmul(Matrix *dA, Matrix *dB) {
    if (dA->device_type != dB->device_type) {
        printf("two Matrix is on different device. dA : %d, dB: %d\n", dA->device_type, dB->device_type);
        return NULL;
    }
    if (dA->col != dB->row) {
        printf("number of column of dA and row of dB doesn't match\n");
        return NULL;
    }
    Matrix *dC = makeMatrix(dA->row, dB->col, dA->device_type);
    hipSetDevice(dA->device_type - 1);
    matmul_(dA->M, dB->M, dC->M, NULL, dA->row, dB->col, dA->col);
    return dC;
}

Matrix *matmul_inline(Matrix *dC, Matrix *dA, Matrix *dB) {
    if (dA->device_type != dB->device_type) {
        printf("two Matrix is on different device. dA : %d, dB: %d\n", dA->device_type, dB->device_type);
        return NULL;
    }
    if (dA->col != dB->row) {
        printf("number of column of dA and row of dB doesn't match\n");
        return NULL;
    }
    if (dC->device_type != dA->device_type) {
        printf("result matrix is on different device.\n");
        return NULL;
    }
    if (dC->row != dA->row || dC->col != dB->col) {
        printf("result matrix is in different dimension.\n");
        return NULL;
    }
    hipSetDevice(dA->device_type - 1);
    matmul_(dA->M, dB->M, dC->M, NULL, dA->row, dB->col, dA->col);
    return dC;
}

__global__ void ReLU_device(float *dA, int size){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(dA[i] < 0){
        dA[i] = 0;
    }
}
Matrix *ReLU_inline(Matrix *mat){//change values directly. doesn't clone.
    int thdsPerBlks = tile_SIZE*tile_SIZE;
    int numofBlks = (mat->row*mat->col+thdsPerBlks-1) / thdsPerBlks;
    hipSetDevice(mat->device_type-1);
    ReLU_device<<<numofBlks, thdsPerBlks>>>(mat->M, mat -> row * mat -> col);
    hipDeviceSynchronize();
    return mat;
}


__global__ void softMax(float*dRes, float *dMat, int row, int col){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < row){
        double sum = 0.0;
        for(int j=0; j < col; j++){
            sum += expf(dMat[i * col + j]);
        }
        for(int j=0; j < col; j++){
            dRes[i * col + j] = expf(dMat[i * col + j]) / sum;
        }
    }
}


Matrix *softMax_Rowwise_inline(Matrix*dRes, Matrix *dMat){
    if(dMat->row == dRes->row && dMat->col == dRes->col){
        softMax<<<((dMat->row + tile_SIZE - 1) / tile_SIZE), tile_SIZE>>>(dRes->M, dMat->M, dMat->row, dMat->col);
        return dRes;
    }else{
        printf("\"softMax_Rowwise_inline\" : src's and res's row and column is not same\n");
        return NULL;
    }
}


int isSameShape(Matrix *dMat1, Matrix *dMat2, Matrix *dMat3){
    return dMat1->row == dMat2->row && dMat1->col == dMat2->col && dMat2->row == dMat3->row && dMat2->col == dMat3-> col && dMat1-> device_type == dMat2 ->device_type &&dMat2->device_type == dMat3->device_type;
}

__global__ void matadd_(float *dMat, float *dA, float *dB, int size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < size){
        dMat[i] = dA[i] + dB[i];
    }
}

Matrix *matAdd(Matrix *dMat, Matrix *dA, Matrix *dB){
    if(isSameShape(dMat, dA, dB)){
        int size = dA->row * dA->col;
        matadd_<<<(size+tile_SIZE*tile_SIZE -1)/tile_SIZE,tile_SIZE * tile_SIZE>>>(dMat->M, dA->M, dB->M, dA->row * dA->col);
        return dMat;
    }else{
        printf("\"matAdd\" : one of dMat, dA, dB's type does not match\n");
        return NULL;
    }
}


__global__ void matsub_(float *dMat, float *dA, float *dB, int size){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < size){
        dMat[i] = dA[i] - dB[i];
    }
}

Matrix *matSub(Matrix *dMat, Matrix *dA, Matrix *dB){
    if(isSameShape(dMat, dA, dB)){
        int size = dA->row * dA->col;
        matsub_<<<(size+tile_SIZE*tile_SIZE -1)/tile_SIZE,tile_SIZE * tile_SIZE>>>(dMat->M, dA->M, dB->M, dA->row * dA->col);
        return dMat;
    }else{
        printf("\"matAdd\" : one of dMat, dA, dB's type does not match\n");
        return NULL;
    }
}

__global__ void eyeMatrix(Matrix*dMat, int inx){

}
// Matrix *eyeMat(Matrix*dMat, int inx){
//     if(dMat->row != 1){
//         printf("row of eyemat should be 1\n");
//         return NULL;
//     }
    
//     return dMat;
// }

// Matrix *softMax_Rowwise_inline(Matrix *res_Mat, Matrix *mat){
//     for(int i=0; i < mat->row; i++){
//         double tmp = 0.0;
//         for(int j=0; j < mat -> col; j++){
//             tmp += exp(mat->M[i * mat->col + j]);
//         }
//         for(int j=0; j < mat -> col; j++){
//             res_Mat->M[i * mat->col + j] = exp(mat->M[i * mat->col + j]) / tmp;
//         }
//     }
//     return res_Mat;
// }

