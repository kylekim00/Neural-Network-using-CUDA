#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include "tensor_struct_2.h"
//나중에는 나눠서 각각의 gpu 안에 넣어야하기 때문에 생각을 해보면 인덱스 값에 따라 값을 copy 해주는 것도 있으면 좋을 것 같다.
//기존 텐서와 다른점. 

//1. dim과 stride가 다를 때를 고려한다. 아울러 총 크기를 그냥 저장해버린다. 이는 makelightcopysubTensor() 함수를 만들기 위함이다. 마치 커서의 드래그와 같은 역할을 해줄 것이다.
//2. num_dim이 1, 2 일때도 작동이 되도록한다.
//3. device 위에 올라가 있을 경우 dim과 stride도 같이 device에 올려준다. [5, 1, 3, 3, 4] X [4, 1, 4, 2] 와 같은 복잡한 텐서도 행렬곱이 가능하게 하기 위함이다. 

Tensor *makeTensor(int *dim, int num_dim, int device_type){
    if(!dim){                       //if There is no dim inside
        return NULL;
    }
    if(num_dim < 0){                //if there is not an appropriate num_dim
        return NULL;
    }
    int device;
    hipGetDeviceCount(&device);
    if(device_type > device){       //count device and check the boundary
        printf("DEVICE NUM %d NOT AVAILABLE\n", device_type);
        return NULL;
    }

    int sizeTensor;                 //check the size of whole Tensor

    Tensor* ten = (Tensor*)malloc(sizeof(Tensor));      //give tensor a space for host
    ten->dim = (int*)malloc(2 * num_dim * sizeof(int)); //give dim and stride a spcae for host
    ten->stride = ten->dim+num_dim;                    //this approach might be effective when sending to GPU later.

    ten->num_dim = num_dim;
    ten->device_type = device_type;

    sizeTensor = 1;
    for(int i= num_dim - 1; i >= 0; i--){
        ten->dim[i] = dim[i];
        ten->stride[i] = sizeTensor;
        sizeTensor *= dim[i];
    }

    ten->sizeTensor = sizeTensor;

    if(!device_type){
        ten->T = (float*)malloc(sizeTensor * sizeof(float));
        ten->d_dim_stride = NULL;
    }else{
        hipSetDevice(device_type-1);
        hipMalloc(&ten->T, sizeTensor * sizeof(float));
        hipMalloc(&ten->d_dim_stride, 2 * num_dim * sizeof(int));
        hipMemcpy(ten->d_dim_stride, ten->dim, 2 * num_dim * sizeof(int), hipMemcpyHostToDevice);
    }
    return ten;
}


Tensor* makeTensorbyShape(Tensor* src, int device_type){
    if(!src){
        printf("SouRCe is vacant.\n");
        return NULL;
    }
    return makeTensor(src->dim, src->num_dim, device_type);
}


//================================================FREEEEEEEEEEEE===============================================================
void freeTensor(Tensor *ten){
    if(ten==NULL){
        printf("NO TENSOR IN POINTER.\n");
        return;
    }else{
        if(ten->device_type){
            hipSetDevice(ten->device_type - 1);
            hipError_t err = hipFree(ten->T);
            if (err != hipSuccess) {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
            }
            err = hipFree(ten->d_dim_stride);
            if (err != hipSuccess) {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
            }
        }else{
            free(ten->T);
        }
    }

    free(ten->dim);         //didn't malloc ten->stride from the first place :P
    free(ten);
}


void infoTensor(Tensor *ten){
    printf("\n=========Tensor===========\n");
    printf("DIMENTION : [");
    for(int i=0; i < ten->num_dim-1; i++){
        printf("%d ", ten->dim[i]);
    }
    printf("%d]\n", ten->dim[ten->num_dim - 1]);
    printf("STRIDE    : [");
    for(int i=0; i < ten->num_dim-1; i++){
        printf("%d ", ten->stride[i]);
    }
    printf("%d]\n", ten->stride[ten->num_dim - 1]);
    printf("DEVICE TYPE : ");
    if(ten->device_type){
        printf("GPU %d", ten->device_type);
    }else{
        printf("CPU");
    }
    printf("\n==========================\n");
}

void printTensor(Tensor *ten){
    if(ten->device_type){
        printf("printTensor : GPU mem can not be printed\n");
        return;
    }
    //==============if ten->num_dim < 3========================
    if(ten->num_dim == 1){
        printf("[ %d ]\n", ten->dim[0]);
        for(int i=0; i < ten->dim[0]; i+=ten->stride[0]){
            printf("%.02f\t", ten->T[i]);
        }
        printf("\n");
        return;
    }
    if(ten->num_dim == 2){
        printf("[ %d %d ]\n", ten->dim[0], ten->dim[1]);
        for(int i=0; i < ten->dim[0]*ten->stride[0]; i+=ten->stride[0]){
            for(int j=0; j < ten->dim[1]*ten->stride[1]; j+=ten->stride[1]){
                printf("%.02f\t", ten->T[i + j]);
                // printf("%d\t", ten->stride[0]* i + j);
            }
            printf("\n");
        }
        printf("\n");
        return;
    }
    //=================else====================================
    printf("=\n");

    int* tmp_Inx = (int*)malloc(sizeof(int) * (ten->num_dim - 2));
    for(int i=0; i < ten->num_dim - 2; i++){
        tmp_Inx[i] = 0;
    }
    int inx;
    while(tmp_Inx[0] < ten->dim[0]){
        inx = 0;
        printf("[ ");
        for(int i=0; i < ten->num_dim-2;i++){
            printf("%d ", tmp_Inx[i]);
            inx += tmp_Inx[i] * ten->stride[i];
        }
        printf("- - ]\n");

        for(int i=0; i < ten->dim[ten->num_dim-2]*ten->stride[ten->num_dim-2]; i+=ten->stride[ten->num_dim-2]){
            for(int j=0; j < ten->dim[ten->num_dim-1]*ten->stride[ten->num_dim-1]; j+=ten->stride[ten->num_dim-1]){
                printf("%.02f\t", ten->T[inx + i + j]);
                // printf("%d\t", ten->stride[0]* i + j);
            }
            printf("\n");
        }

        tmp_Inx[ten->num_dim - 3]++;
        for(int i = ten->num_dim - 3; i > 0; i--){
            if(tmp_Inx[i] >= ten->dim[i]){
                tmp_Inx[i-1]++;
                tmp_Inx[i] = 0;
            }
        }
    }
}

Tensor* copyTensor(Tensor *dst, Tensor *src){
    if(dst->num_dim != src->num_dim){
        printf("copyMatrix : shape of dst and src doesn't match.\n");
        return NULL;
    }
    for(int i=0; i < dst->num_dim; i++){
        if(dst->dim[i] != src->dim[i]){
            printf("copyMatrix : shape of dst and src doesn't match.\n");
            return NULL;
        }
    }
    if(!dst->device_type && !src->device_type){ //CPU to CPU
        for(int i=0; i < dst->dim[0]*dst->stride[0]; i++)
            dst->T[i] = src->T[i];
    }

    else if(dst->device_type && src->device_type){
        hipMemcpy(dst->T, src->T, dst->dim[0]*dst->stride[0] * sizeof(float), hipMemcpyDeviceToDevice);
    }
    else if(dst->device_type){
        hipSetDevice(dst->device_type -1);
        hipMemcpy(dst->T, src->T, dst->dim[0]*dst->stride[0] * sizeof(float), hipMemcpyHostToDevice);
    }else{
        hipSetDevice(src->device_type -1);
        hipMemcpy(dst->T, src->T, dst->dim[0]*dst->stride[0] * sizeof(float), hipMemcpyDeviceToHost);
    }
    return dst;
}

Tensor* makelightcopysubTensor(Tensor* src){
}


__global__ void reshape_(float* dst, float* src){
    
}


Tensor* reshapeTensor(Tensor* dst, Tensor* src, int* reshape){

    if(src->device_type != dst->device_type){
        printf("DEVICE NOT MATCH.\n");
        return NULL;
    }
    if(src->num_dim != dst->num_dim){
        printf("DEVICE NUM_DIM DOES NOT MATCH.\n");
        return NULL;
    }

    if(src->dim[0] * src->stride[0] != dst->dim[0] * dst->stride[0]){
        printf("DEVICE NUM OF ELEMENT DOES NOT MATCH.\n");
        return NULL;
    }

    //===================Setting for reshape==========================
    int* tmp_reshape = (int*)malloc(sizeof(int) * src->num_dim);
    for(int i=0; i < src->num_dim; i++){
        printf("%d ", reshape[i]);
    }
    printf("\n");
    for(int i=0; i < src->num_dim; i++){
        for(int j=0; j < src->num_dim; j++){
            if(reshape[j] == i && dst->dim[j] == src->dim[i]){//여기서 reshape이랑 맞지 않는 것도 걸러냄.
                tmp_reshape[i] = j;
                goto NEXT_RESHAPETENSOR_TMP_RESHAPE;//나도 쓰기 싫었다.
            }
        }
        printf("NOT AN APPROPRIATE RESHAPE.\n");
        return NULL;
        NEXT_RESHAPETENSOR_TMP_RESHAPE: ;
    }
    //================================================================

    for(int i=0; i < src->num_dim; i++){
        printf("%d ", tmp_reshape[i]);
    }

    printf("\n");
    if(src->device_type){//GPU
        hipSetDevice(src->device_type - 1);
        
        
    }else{//CPU
        int newInx, tmp;
        for(int inx =0; inx < src->sizeTensor; inx++){
            newInx = 0;
            tmp = inx;
            for(int i=0; i < src->num_dim; i++){
                newInx += tmp / src->stride[i] * dst->stride[tmp_reshape[i]];
                tmp = tmp % src->stride[i];
            }
            dst->T[newInx] = src->T[inx];
        }
    }

    free(tmp_reshape);
    return dst;
}



// __global__ void tiledMM(float *A, float *B, float *C, float *bias, int M, int N, int K) {
//     //blockDim.z, blockIdx.z 이게 매트릭스의 수//z thread를 늘린다고 문제가 해결되지 않는다.
    
//     int matIdx = blockDim.z * blockIdx.z;
//     int row = blockDim.y * blockIdx.y + threadIdx.y;
//     int col = blockDim.x * blockIdx.x + threadIdx.x;
//     __shared__ float s_a[tile_SIZE][tile_SIZE];
//     __shared__ float s_b[tile_SIZE][tile_SIZE];

//     float tmp = 0.0f;

//     for (int i = 0; i < (K + tile_SIZE - 1) / tile_SIZE; i++) {
//         if (row < M && (i * tile_SIZE + threadIdx.x) < K)
//         //A와 B에 데이터를 넣을 때 matIdx에 차이를 두어야 한다.
//         //예를 들자면 matIdxA = blockDim.z * blockIdx.z / big_Dim_stride; 이러면 반복이 되니까.
//             s_a[threadIdx.y][threadIdx.x] = A[matIdx * (K*M) + row * K + (i * tile_SIZE + threadIdx.x)];
//         else
//             s_a[threadIdx.y][threadIdx.x] = 0.0f;

//         if (col < N && (i * tile_SIZE + threadIdx.y) < K)
//             s_b[threadIdx.y][threadIdx.x] = B[matIdx * (K*N) +(i * tile_SIZE + threadIdx.y) * N + col];
//         else
//             s_b[threadIdx.y][threadIdx.x] = 0.0f;

//         __syncthreads();

//         for (int j = 0; j < tile_SIZE; j++) {
//             tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
//         }

//         __syncthreads();
//     }

//     if (row < M && col < N) {
//         if (bias)
//             C[matIdx *(M*N) + row * N + col] = tmp + bias[col];
//         else
//             C[matIdx *(M*N) + row * N + col] = tmp;
//     }
// }

__global__ void tiledMM(float *A, float *B, float *C, float *bias, int M, int N, int K, int big_dim_stride, int big_A_True) {
    //blockDim.z, blockIdx.z 이게 매트릭스의 수//z thread를 늘린다고 문제가 해결되지 않는다.
    int matIdx_A, matIdx_B;
    if(big_A_True){
        matIdx_A = blockDim.z * blockIdx.z;
        matIdx_B = matIdx_A % big_dim_stride;
    }else{
        matIdx_B = blockDim.z * blockDim.z;
        matIdx_A = matIdx_B % big_dim_stride;
    }
    
    
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float s_a[tile_SIZE][tile_SIZE];
    __shared__ float s_b[tile_SIZE][tile_SIZE];

    float tmp = 0.0f;

    for (int i = 0; i < (K + tile_SIZE - 1) / tile_SIZE; i++) {
        if (row < M && (i * tile_SIZE + threadIdx.x) < K)
        //A와 B에 데이터를 넣을 때 matIdx에 차이를 두어야 한다.
        //예를 들자면 matIdxA = blockDim.z * blockIdx.z / big_Dim_stride; 이러면 반복이 되니까.
            s_a[threadIdx.y][threadIdx.x] = A[matIdx_A * (K*M) + row * K + (i * tile_SIZE + threadIdx.x)];
        else
            s_a[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && (i * tile_SIZE + threadIdx.y) < K)
            s_b[threadIdx.y][threadIdx.x] = B[matIdx_B * (K*N) +(i * tile_SIZE + threadIdx.y) * N + col];
        else
            s_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int j = 0; j < tile_SIZE; j++) {
            tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        if (bias)
            tmp = tmp + bias[col];
        if(big_A_True)
            C[matIdx_A *(M*N) + row * N + col] = tmp;
        else
            C[matIdx_B *(M*N) + row * N + col] = tmp;
    }
}

void matmul_matwise_(float *dA, float *dB, float *dC, float *dBias, int M, int N, int K, int numofMat, int big_dim_stride, int big_A_True) {
    dim3 dimGrid((N + tile_SIZE - 1) / tile_SIZE, (M + tile_SIZE - 1) / tile_SIZE, numofMat); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
    dim3 dimBlock(tile_SIZE, tile_SIZE);
    tiledMM<<<dimGrid, dimBlock>>>(dA, dB, dC, dBias, M, N, K, big_dim_stride, big_A_True);
}

Tensor* matmul_matwise(Tensor*dC, Tensor *dA, Tensor *dB){
    if (dA->device_type != dB->device_type) {
        printf("two source Matrix is on different device. dA : %d, dB: %d\n", dA->device_type, dB->device_type);
        return NULL;
    }
    if (dC->device_type != dA->device_type) {
        printf("result matrix is on different device.dA : %d, dC: %d\n", dA->device_type, dC->device_type);
        return NULL;
    }

    if(dA->dim[dA->num_dim - 1] != dB->dim[dB->num_dim-2]){//dA와 dB의 row, col이 맞는지를 확인하는 작업.
        printf("number of column of dA and row of dB doesn't match\n");
        return NULL;
    }

    int big_A_True = 0;//이건 어느게 더 큰놈인지 판단을 하는 것. A가 크면 1
    
    Tensor* bigTensor, *smallTensor;//큰놈을 큰놈에, 작은놈을 작은놈에
    if(dA->num_dim > dB->num_dim){
        big_A_True = 1;
        bigTensor = dA;
        smallTensor = dB;
    }else{
        big_A_True = 0;
        bigTensor = dB;
        smallTensor = dA;
    }
    
    int dim_contrast = bigTensor->num_dim - smallTensor->num_dim;

    for(int i= smallTensor->num_dim - 3; i >= 0; i--){// bigTensor와 smallTensor의 차원을 서로 비교하는 것.
        if(bigTensor->dim[i + dim_contrast] != smallTensor-> dim[i]){
            printf("matrices have different dimension.\n");
            return NULL;
        }
    }

    if(dC->num_dim != bigTensor->num_dim){
        printf("matrices have different dimension.(num_dim is different)\n");
        return NULL;
    }

    for(int i=0; i < bigTensor->num_dim-3; i++){
        if(dA->dim[i] != dC->dim[i]){
            printf("dim %d of source and result Matrix is not the same.\n", i);
            return NULL;
        }
    }
    if(dA->dim[dA->num_dim - 2] != dC->dim[dA->num_dim - 2] || dC->dim[dC->num_dim -1] != dB->dim[dB->num_dim - 1]){
        printf("result matrix is in different dimension.\n");
        return NULL;
    }
    int big_dim_stride = bigTensor->stride[dim_contrast-1] / bigTensor->stride[bigTensor->num_dim-3];
    matmul_matwise_(dA->T, dB->T, dC->T, NULL, dA->dim[dA->num_dim - 2], dB->dim[dB->num_dim - 1], dA->dim[dA->num_dim - 1], bigTensor->dim[0] * bigTensor->stride[0] / bigTensor->stride[bigTensor->num_dim-3], big_dim_stride, big_A_True);
    return dC;

}

