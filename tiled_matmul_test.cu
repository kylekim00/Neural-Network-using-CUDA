//nvcc  tiled_matmul_test.cu matrix_struct.cu -I.
#include "matrix_struct.h"
#include<unistd.h>
#include<hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define cudaCheckError(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

Matrix* dummyMatrix(Matrix *mat){
    for(int i=0; i < mat->row * mat-> col; i++){
        float dm = i;
        mat->M[i] = dm;
    }
    return mat;
}
void zeroMatrix(Matrix *mat){
    for(int i=0; i < mat->row * mat-> col; i++){
        mat->M[i] = 0;
    }
}
int main(){
    // //Memory Test
    // Matrix *A = makeMatrix(10, 10, 0);
    // for(int i=0; i < 10; i++){
    //     A = moveMatrix(A, 1);//사용하는 GPU 메모리의 크기가 커지지 않는다.
    //     // sleep(5);
    //     A = moveMatrix(A, 0);
    // }

    Matrix *A = makeMatrix(5, 7, 0);
    dummyMatrix(A);
    Matrix *B = copyMatrix(makeMatrix(5, 7, 1), A);
    infoMatrix(B);
    Matrix *C = copyMatrix(makeMatrix(5, 7, 2), B);
    A = copyMatrix(A, C);
    printMatrix(A);
    printMatrix(copyMatrix(makeMatrix(B->row, B->col, 0), transposeMatrix(B)));

    cudaCheckError(hipGetLastError());
    cudaCheckError(hipDeviceSynchronize());
}

//쿠다를 해제한다고 해서 할당된 메모리가 그냥 의미없이 사라지는 것이 아니다. 마치 휴지통에 지운다고 해서
//그 메모리를 못쓰는게 아닌것 처럼. 그냥 덮어서 쓸 뿐이다. 그래서 free를 시켜도 할당된 메모리 사용랑이 변하지 않는 것이다. 
//걍 더 쓸 때 더 늘어날 뿐이다.
