#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include"./Easy_Tensor/easy_tensor.h"
#include<string.h>
Tensor* dummyTensor(Tensor *ten){
    for(int i=0; i < ten->dim[0] * ten->stride[0]; i++){
        ten->T[i] = __FLT_MAX__/i;
    }
    return ten;
}
__global__ void CESoftmax_deriv_(float* deriv, float* O, float* label, int O_stride, int batch_size){
    int inx = blockDim.x * blockIdx.x + threadIdx.x;//each batch
    for(int i=0; i < O_stride;i++){
        if(i == label[inx]-1)
            deriv[O_stride * inx + i] = O[O_stride * inx + i] - 1;
        else
            deriv[O_stride * inx + i] = O[O_stride * inx + i];
    }
}
Tensor* CESoftmax_deriv(Tensor* d_der_O, Tensor*d_O, Tensor* d_Y){
    if(!d_der_O || !d_O || !d_Y){
        printf("CES: no Tensor.\n");
        return NULL;
    }
    if(d_O->dim[0] != d_Y->sizeTensor){
        printf("batch does not match.\n");
        return NULL;
    }
    if(d_der_O->num_dim != d_O->num_dim){
        printf("dimention does not match.\n");
        return NULL;
    }
    hipSetDevice(d_der_O->device_type-1);
    CESoftmax_deriv_<<<(d_der_O->dim[d_der_O->num_dim - 2] + tile_SIZE - 1)/tile_SIZE, tile_SIZE>>>(d_der_O->T,d_O->T, d_Y->T, d_der_O->dim[d_der_O->num_dim - 1], d_der_O->dim[d_der_O->num_dim - 2]);
    return d_der_O;
}

Tensor* dummyTensor2(Tensor *ten){
    for(int i=0; i < ten->dim[0] * ten->stride[0]; i++){
        ten->T[i] = i+1;
    }
    return ten;
}
int main(){
    // Tensor *A = makeTensor("3 196 768", 0);
    // A = dummyTensor(A);
    // Tensor* B = dummyTensor(makeTensor("1 768 2304", 0));
    // // printTensor(ReLU_inline(B));
    // Tensor*bias = makeTensor("2304", 0);
    // for(int i=0; i < bias->sizeTensor; i++)
    //     bias->T[i] = i;
    
    // Tensor *dA = copyTensor(makeTensorbyShape(A, 1), A);
    // Tensor* dB = copyTensor(makeTensorbyShape(B, 1),B);

    // Tensor *dC = matmul_bias(makeTensor("3 196 2304",1), dA, dB, copyTensor(makeTensorbyShape(bias, 1), bias), 0);
    // // Tensor *dC = matmul(makeTensorbyShape(dB, 1), dA, dB);

    // printTensor(makeSubTensor(copyTensor(makeTensorbyShape(dC, 0), dC),"0 0 2296", "8 8"));
    ///////////////////////////////////////////////////////////////////////////////////////
    Tensor* A = dummyTensor(makeTensor("1 32", 0));
    Tensor* O = makeTensorbyShape(A, 0);
    Tensor* dA = copyTensor(makeTensorbyShape(A, 1), A);
    Tensor* dO = makeTensorbyShape(dA, 1);
    softMax(dO, dA);
    softMax(O, A);
    printTensor(O);

    printTensor(copyTensor(A, O));

    // Tensor* B = dummyTensor(makeTensor("16 40", 0));
    // Tensor* dB = copyTensor(makeTensorbyShape(B, 1), B);
    // Tensor* dAt = makeTensor("2 3 5 3", 1);
    // Tensor* At = makeTensor("2 2 5 3", 0);
    // Tensor* C = makeTensor("10 40", 0);
    // Tensor* dC = makeTensorbyShape(C, 1);

    
    // dA = scalar_Tensor(dA, '-', 0);
    // elementWise_Tensor(dA, dA, '*', dB); 

    // freeTensor(printTensor(copyTensor(At,copyTransposeTensor(dAt, dA))));


    // freeTensor(printTensor(copyTensor(makeTensor("16", 0), rowcolwise_sum(makeTensor("16", 1), dA, 1))));


    // Tensor* subA = makeSubTensor(dA, "0 0 0", "3 5 5");
    // printTensor(copyTensor(makeTensorbyShape(dB, 0),copyTransposeTensor(makeTensorbyShape(dB, 1),dB)));
    // printTensor(copyTensor(makeTensor("3 4 5 5", 0),matmul(makeTensor("3 4 5 5", 1), dA, dB));
    // Tensor* subA = makeSubTensor(A, "0 0 0 1", "3 1 2 2");
    
    // Tensor* dA = copyTensor(makeTensorbyShape(A, 1), A);
    // Tensor* dB = copyTensor(makeTensorbyShape(B, 1), B);
    // Tensor *dA_sub = makeSubTensor(dA, "0 0 0 1", "3 1 2 2");
    // int df[] = {0, 1, 2, 3};
    // printTensor(copyTensor(makeTensorbyShape(dA_sub, 0),copyReshapeTensor(makeTensorbyShape(dA_sub, 1), dA_sub, df)));
    // printTensor(B);
    // Tensor* dC = compMatMul(makeTensor("3, 3, 5, 2", 1), dB, dA_sub);
    // printTensor(copyTensor(makeTensorbyShape(dC, 0), dC));


    // Tensor*dAt = makeTensor("3, 2, 4, 4", 1);
    // Tensor*At = makeTensorbyShape(dAt, 0);
    // printf("dAt : %d", dAt->sizeTensor);
    // int df[] = {1, 2, 0, 3};
    // copyReshapeTensor(dAt, dA, df);
    // copyTensor(At, dAt);
    // printTensor(At);
    // freeTensor(At);

    // for(int i=0; i < 4; i++){
    //     df[i] = i;
    // }
    // Tensor* dsubA = makeSubTensor(dA, "0 0 0 1", "3 1 2 2");
    // printTensor(copyTensor(makeTensorbyShape(dsubA, 0),copyReshapeTensor(makeTensorbyShape(dsubA, 1), dsubA, df)));
    // infoTensor(dsubA);
    
    // Tensor* dC = compMatMul(makeTensor("3 3 2 2", 1), dsubA, copyTensor(makeTensorbyShape(B, 1), B));

    // printTensor(copyTensor(makeTensorbyShape(dC, 0), dC));

    


    // printTensor(A);
    // printTensor(subA);
    // freeTensor(subA);
}