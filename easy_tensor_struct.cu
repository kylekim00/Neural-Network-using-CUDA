#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include <math.h>

#include "easy_tensor.h"
//나중에는 나눠서 각각의 gpu 안에 넣어야하기 때문에 생각을 해보면 인덱스 값에 따라 값을 copy 해주는 것도 있으면 좋을 것 같다.
//기존 텐서와 다른점. 

//1. dim과 stride가 다를 때를 고려한다. 아울러 총 크기를 그냥 저장해버린다. 이는 makelightcopysubTensor() 함수를 만들기 위함이다. 마치 커서의 드래그와 같은 역할을 해줄 것이다.(해결)
//2. num_dim이 1, 2 일때도 작동이 되도록한다.
//3. device 위에 올라가 있을 경우 dim과 stride도 같이 device에 올려준다. [5, 1, 3, 3, 4] X [4, 1, 4, 2] 와 같은 복잡한 텐서도 행렬곱이 가능하게 하기 위함이다. 



Tensor *mallocTensor(int *dim, int num_dim, int device_type){
    if(!dim){                       //if There is no dim inside
        return NULL;
    }
    if(num_dim < 0){                //if there is not an appropriate num_dim
        return NULL;
    }
    int device;
    hipGetDeviceCount(&device);
    if(device_type > device){       //count device and check the boundary
        printf("DEVICE NUM %d NOT AVAILABLE\n", device_type);
        return NULL;
    }

    int sizeTensor;                 //check the size of whole Tensor

    Tensor* ten = (Tensor*)malloc(sizeof(Tensor));      //give tensor a space for host
    ten->dim = (int*)malloc(2 * num_dim * sizeof(int)); //give dim and stride a spcae for host
    ten->stride = ten->dim+num_dim;                    //this approach might be effective when sending to GPU later.

    ten->num_dim = num_dim;
    ten->device_type = device_type;

    sizeTensor = 1;
    for(int i= num_dim - 1; i >= 0; i--){
        ten->dim[i] = dim[i];
        ten->stride[i] = sizeTensor;
        sizeTensor *= dim[i];
    }

    ten->sizeTensor = sizeTensor;

    if(!device_type){
        ten->T = (float*)malloc(sizeTensor * sizeof(float));
        ten->d_dim_stride = NULL;
    }else{
        hipSetDevice(device_type-1);
        hipMalloc(&ten->T, sizeTensor * sizeof(float));
        hipMalloc(&ten->d_dim_stride, 2 * num_dim * sizeof(int));
        hipMemcpy(ten->d_dim_stride, ten->dim, 2 * num_dim * sizeof(int), hipMemcpyHostToDevice);
    }
    ten->isSub = 0;
    return ten;
}

Tensor* makeTensor(const char dim[], int device_type) {  // Use `const char[]`
    int dim_[MAX_NUM_DIM];  // Array to store dimensions
    int num_dim = 0;        // Counter for the number of dimensions

    const char *ptr = dim;  // Pointer to traverse the string (now `const`)

    // Skip leading spaces
    while (*ptr == ' ') {
        ptr++;
    }

    // Parse the dimension string
    while (*ptr != '\0' && num_dim < MAX_NUM_DIM) {  // Ensure we don't exceed max dims
        // Skip spaces and commas between numbers
        while (*ptr == ' ' || *ptr == ',') {
            ptr++;
        }

        if (*ptr == '\0') {
            break;  // End of string
        }

        int value = 0;
        int sign = 1;

        // Optional: Handle negative numbers
        if (*ptr == '-') {
            sign = -1;
            ptr++;
        }

        // Convert digit characters to integer
        while (*ptr >= '0' && *ptr <= '9') {
            value = value * 10 + (*ptr - '0');
            ptr++;
        }

        // Store the parsed number in the dimensions array
        dim_[num_dim++] = sign * value;

        // Skip any spaces or commas after the number
        while (*ptr == ' ' || *ptr == ',') {
            ptr++;
        }
    }

    // If no dimensions were parsed, return NULL or handle the error appropriately
    if (num_dim == 0) {
        printf("No valid dimensions were parsed.\n");
        return NULL;
    }

    // Call makeTensor with the parsed dimensions
    return mallocTensor(dim_, num_dim, device_type);
}


Tensor* makeTensorbyShape(Tensor* src, int device_type){
    if(!src){
        printf("SouRCe is vacant.\n");
        return NULL;
    }
    // if(src->isSub){
    //     printf("Source is SubTensor.\n");
    //     return NULL;
    // }
    return mallocTensor(src->dim, src->num_dim, device_type);
}


Tensor* mallocSubTensor(Tensor* src, int* start_point, int* dim, int num_dim){
    if(src->isSub){
        printf("Cant light copy subTensor\n");
        return NULL;
    }
    if(src->num_dim < num_dim){
        printf("SouRCe num_dim not that big\n");
        return NULL;
    }

    int cont = src->num_dim - num_dim;
    float* sp = src->T;


    for(int i=0; i < src->num_dim; i++){                            //This is where you set the starting point
        if(src->dim[i] <= start_point[i]){
            printf("starting point invalid\n");
            return NULL;
        }
        sp += start_point[i] * src->stride[i];
    }

    for(int i=0; i < num_dim; i++){                                 //This is where tou check the size of the dim 
        if(src->dim[i + cont] < start_point[i+cont] + dim[i]){
            printf("SouRCe not that big.\n");
            return NULL;
        }
    }

    Tensor* subTensor = (Tensor*)malloc(sizeof(Tensor));            //Tensor malloc

    subTensor->isSub = 1;
    subTensor->device_type = src->device_type;                      //device_type same as src
    subTensor->num_dim = num_dim;                                   //num_dim

    subTensor->dim = (int*)malloc(2 * sizeof(int)* num_dim);        //dim stride malloc
    subTensor->stride = subTensor->dim + num_dim;

    subTensor->sizeTensor = 1;
    for(int i=0; i < num_dim; i++){                             
        subTensor->dim[i] = dim[i];
        subTensor-> stride[i] = src->stride[i+cont];            //copy Stride
        subTensor->sizeTensor *= dim[i];
    }

    subTensor->T = sp;

    if(src->device_type){
        hipSetDevice(src->device_type-1);
        hipMalloc(&subTensor->d_dim_stride, 2 * num_dim * sizeof(int));
        hipMemcpy(subTensor->d_dim_stride, subTensor->dim, 2 * num_dim * sizeof(int), hipMemcpyHostToDevice);
    }else{
        subTensor->d_dim_stride = NULL;
    }

    return subTensor;
}

Tensor* makeSubTensor(Tensor* src, const char start_point[], const char dim[]){

    int start_point_[MAX_NUM_DIM];  // Array to store dimensions
    int num_sp = 0;        // Counter for the number of dimensions

    const char *ptr = start_point;  // Pointer to traverse the string (now `const`)

    // Skip leading spaces
    while (*ptr == ' ') {
        ptr++;
    }

    // Parse the dimension string
    while (*ptr != '\0') {  // Ensure we don't exceed max dims
        // Skip spaces and commas between numbers
        while (*ptr == ' ' || *ptr == ',') {
            ptr++;
        }

        if (*ptr == '\0') {
            break;  // End of string
        }

        int value = 0;
        int sign = 1;

        // Optional: Handle negative numbers
        if (*ptr == '-') {
            sign = -1;
            ptr++;
        }

        // Convert digit characters to integer
        while (*ptr >= '0' && *ptr <= '9') {
            value = value * 10 + (*ptr - '0');
            ptr++;
        }

        // Store the parsed number in the dimensions array
        start_point_[num_sp++] = sign * value;

        // Skip any spaces or commas after the number
        while (*ptr == ' ' || *ptr == ',') {
            ptr++;
        }
    }

    // If no dimensions were parsed, return NULL or handle the error appropriately
    if (num_sp != src->num_dim) {
        printf("not an appropriate dimention number for starting point");
        return NULL;
    }

    ///////////////////////////////////////////////////////////////////////
    int dim_[MAX_NUM_DIM];  // Array to store dimensions
    int num_dim = 0;        // Counter for the number of dimensions

    ptr = dim;  // Pointer to traverse the string (now `const`)

    // Skip leading spaces
    while (*ptr == ' ') {
        ptr++;
    }

    // Parse the dimension string
    while (*ptr != '\0' && num_dim < MAX_NUM_DIM) {  // Ensure we don't exceed max dims
        // Skip spaces and commas between numbers
        while (*ptr == ' ' || *ptr == ',') {
            ptr++;
        }

        if (*ptr == '\0') {
            break;  // End of string
        }

        int value = 0;
        int sign = 1;

        // Optional: Handle negative numbers
        if (*ptr == '-') {
            sign = -1;
            ptr++;
        }

        // Convert digit characters to integer
        while (*ptr >= '0' && *ptr <= '9') {
            value = value * 10 + (*ptr - '0');
            ptr++;
        }

        // Store the parsed number in the dimensions array
        dim_[num_dim++] = sign * value;

        // Skip any spaces or commas after the number
        while (*ptr == ' ' || *ptr == ',') {
            ptr++;
        }
    }

    // If no dimensions were parsed, return NULL or handle the error appropriately
    if (num_dim == 0) {
        printf("No valid dimensions were parsed.\n");
        return NULL;
    }

    return mallocSubTensor(src, start_point_, dim_, num_dim);
}

//================================================FREEEEEEEEEEEE===============================================================

void freeSubTensor(Tensor* subTen){
    if(!subTen->isSub){
        printf("This is Not a SubTensor.\n");
        return;
    }
    if(subTen==NULL){
        printf("NO TENSOR IN POINTER.\n");
        return;
    }else{
        if(subTen->device_type){
            hipSetDevice(subTen->device_type - 1);
            hipError_t err = hipFree(subTen->d_dim_stride);
            if (err != hipSuccess) {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
            }
        }
        free(subTen->dim);         //didn't malloc ten->stride from the first place :P
        free(subTen);
    }
}
void freeTensor(Tensor *ten){
    if(ten==NULL){
        printf("NO TENSOR IN POINTER.\n");
        return;
    }
    if(ten->isSub){
        freeSubTensor(ten);
        return;
    }else{
        if(ten->device_type){
            hipSetDevice(ten->device_type - 1);
            hipError_t err = hipFree(ten->T);
            if (err != hipSuccess) {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
            }
            err = hipFree(ten->d_dim_stride);
            if (err != hipSuccess) {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
            }
        }else{
            free(ten->T);
        }
    }
    free(ten->dim);         //didn't malloc ten->stride from the first place :P
    free(ten);
}

//=================================print================================

void infoTensor(Tensor *ten){
    printf("\n=========Tensor===========\n");
    if(ten->isSub){
        printf("===SUBTENSOR===\n");
    }
    printf("DIMENTION : [");
    for(int i=0; i < ten->num_dim-1; i++){
        printf("%d ", ten->dim[i]);
    }
    printf("%d]\n", ten->dim[ten->num_dim - 1]);
    printf("STRIDE    : [");
    for(int i=0; i < ten->num_dim-1; i++){
        printf("%d ", ten->stride[i]);
    }
    printf("%d]\n", ten->stride[ten->num_dim - 1]);
    printf("DEVICE TYPE : ");
    if(ten->device_type){
        printf("GPU %d", ten->device_type);
    }else{
        printf("CPU");
    }
    printf("\n==========================\n");
}

Tensor* printTensor(Tensor *ten){
    if(!ten){
        printf("No tensor.\n");
        return NULL;
    }
    if(ten->device_type){
        printf("printTensor : GPU mem can not be printed\n");
        return ten;
    }
    infoTensor(ten);
    //==============if ten->num_dim < 3========================
    if(ten->num_dim == 1){
        printf("[ %d ]\n", ten->dim[0]);
        for(int i=0; i < ten->dim[0]; i+=ten->stride[0]){
            printf("%.02f\t", ten->T[i]);
        }
        printf("\n");
        return ten;
    }
    if(ten->num_dim == 2){
        printf("[ %d %d ]\n", ten->dim[0], ten->dim[1]);
        for(int i=0; i < ten->dim[0]*ten->stride[0]; i+=ten->stride[0]){
            for(int j=0; j < ten->dim[1]*ten->stride[1]; j+=ten->stride[1]){
                printf("%.02f\t", ten->T[i + j]);
                // printf("%d\t", ten->stride[0]* i + j);
            }
            printf("\n");
        }
        printf("\n");
        return ten;
    }
    //=================else====================================
    printf("=\n");

    int* tmp_Inx = (int*)malloc(sizeof(int) * (ten->num_dim - 2));
    for(int i=0; i < ten->num_dim - 2; i++){
        tmp_Inx[i] = 0;
    }
    int inx;
    while(tmp_Inx[0] < ten->dim[0]){
        inx = 0;
        printf("[ ");
        for(int i=0; i < ten->num_dim-2;i++){
            printf("%d ", tmp_Inx[i]);
            inx += tmp_Inx[i] * ten->stride[i];
        }
        printf("- - ]\n");

        for(int i=0; i < ten->dim[ten->num_dim-2]*ten->stride[ten->num_dim-2]; i+=ten->stride[ten->num_dim-2]){
            for(int j=0; j < ten->dim[ten->num_dim-1]*ten->stride[ten->num_dim-1]; j+=ten->stride[ten->num_dim-1]){
                printf("%.02f\t", ten->T[inx + i + j]);
                // printf("%d\t", ten->stride[0]* i + j);
            }
            printf("\n");
        }

        tmp_Inx[ten->num_dim - 3]++;
        for(int i = ten->num_dim - 3; i > 0; i--){
            if(tmp_Inx[i] >= ten->dim[i]){
                tmp_Inx[i-1]++;
                tmp_Inx[i] = 0;
            }
        }
    }
    printf("=\n");
    free(tmp_Inx);
    return ten;
}

Tensor* copyTensor(Tensor *dst, Tensor *src){
    if(!dst || !src){
        printf("No dst or src\n");
        return NULL;
    }
    if(dst->isSub||src->isSub){
        printf("dst or src is subTensor.");
        return NULL;
    }
    if(dst->num_dim != src->num_dim){
        printf("copyMatrix : shape of dst and src doesn't match.\n");
        return NULL;
    }
    for(int i=0; i < dst->num_dim; i++){
        if(dst->dim[i] != src->dim[i]){
            printf("copyMatrix : shape of dst and src doesn't match.\n");
            return NULL;
        }
    }
    if(!dst->device_type && !src->device_type){ //CPU to CPU
        for(int i=0; i < dst->dim[0]*dst->stride[0]; i++)
            dst->T[i] = src->T[i];
    }

    else if(dst->device_type && src->device_type){
        hipMemcpy(dst->T, src->T, dst->sizeTensor * sizeof(float), hipMemcpyDeviceToDevice);
    }
    else if(dst->device_type){
        hipSetDevice(dst->device_type -1);
        hipMemcpy(dst->T, src->T, dst->sizeTensor * sizeof(float), hipMemcpyHostToDevice);
    }else{
        hipSetDevice(src->device_type -1);
        hipMemcpy(dst->T, src->T, dst->sizeTensor * sizeof(float), hipMemcpyDeviceToHost);
    }
    return dst;
}


__global__ void reshape_(float* dst, float* src, int* dst_dim_stride, int* src_dim_stride, int* reshape, int num_dim, int sizeTensor){
    src_dim_stride += num_dim;
    dst_dim_stride += num_dim;

    int new_inx = blockDim.x * blockIdx.x + threadIdx.x;
    int new_tmp = new_inx;
    int inx = 0;
    for(int i=0; i < num_dim; i++){
        inx += new_tmp / dst_dim_stride[i] * src_dim_stride[reshape[i]];
        new_tmp %= dst_dim_stride[i];
    }
    if(new_inx < sizeTensor)                            //This is for dst because tile size could go over the sizeTensor.
        dst[new_inx] = src[inx];


}


Tensor* copyReshapeTensor(Tensor* dst, Tensor* src, int* reshape){
    if(dst->isSub){
        printf("dst can't be subMatrix.\n");
        return NULL;
    }
    if(src->device_type != dst->device_type){
        printf("DEVICE NOT MATCH.\n");
        return NULL;
    }
    if(src->num_dim != dst->num_dim){
        printf("DEVICE NUM_DIM DOES NOT MATCH.\n");
        return NULL;
    }

    if(src->sizeTensor != dst->sizeTensor){
        printf("DEVICE NUM OF ELEMENT DOES NOT MATCH.\n");
        return NULL;
    }

    if(src->device_type){//GPU
        hipSetDevice(src->device_type - 1);
        int* d_tmp_reshape;
        hipMalloc(&d_tmp_reshape, sizeof(int) * dst->num_dim);
        hipMemcpy(d_tmp_reshape, reshape, sizeof(int) * dst->num_dim, hipMemcpyHostToDevice);
        
        int s_tile_SIZE = tile_SIZE * tile_SIZE * 2;//no special drawbacks in parallel sequence 임
        
        reshape_<<< (dst->sizeTensor + s_tile_SIZE - 1)/s_tile_SIZE, s_tile_SIZE >>>(dst->T, src->T, dst->d_dim_stride, src->d_dim_stride, d_tmp_reshape, src->num_dim, dst->sizeTensor);
        hipFree(d_tmp_reshape);

    }else{//CPU
        int newInx, inx;
        for(int new_inx =0; new_inx < dst->sizeTensor; new_inx++){
            newInx = new_inx;
            inx = 0;
            for(int i=0; i < src->num_dim; i++){
                inx += newInx / dst->stride[i] * src->stride[reshape[i]];
                newInx = newInx % dst->stride[i];
            }
            dst->T[new_inx] = src->T[inx];
        }
    }

    return dst;
}


//빠르긴 한데....
__global__ void transposeCoalesced(float *odata, const float *idata, int dst_row, int dst_col, int src_col_stride, int dst_mat_stride, int src_mat_stride){
    __shared__ float tile[tile_SIZE][tile_SIZE];
    int x = blockIdx.y * tile_SIZE + threadIdx.x;  // 이건 dst를 기준으로 만든것. 일단 dim은 똑같기 때문에 조심해서 해보자. 
    int y = blockIdx.x * tile_SIZE + threadIdx.y;
    int z = blockIdx.z;
    if(x < dst_row)
        for (int j = 0; j < tile_SIZE && (y+j) < dst_col; j++)
            tile[threadIdx.x][threadIdx.y + j] = idata[z * src_mat_stride + (y+j)*src_col_stride + x];//src_row는 stride[num_dim - 2] 그리고 어차피 필요한건 다 저장이 되기 때문에 굳이 쓰레긱 값에 집중하지 말자.
    __syncthreads();
    x = blockIdx.x * tile_SIZE + threadIdx.x;
    y = blockIdx.y * tile_SIZE + threadIdx.y;

    // if(x ==8 && y==0 && z==0){
    //     for(int i=0; i < tile_SIZE; i++){
    //         for(int j=0; j < tile_SIZE; j++){
    //             printf("%.02f ", tile[i][j]);
    //         }
    //         printf("\n");
    //     }
    // }
    
    if(x < dst_col){
        for (int j = 0; j < tile_SIZE && y+j < dst_row; j++)
            odata[z * dst_mat_stride + (y+j) * dst_col + x] = tile[threadIdx.y+j][threadIdx.x];
    }
}

//////////////////////////////////////////////////////////////////////////////////

Tensor* copyTransposeTensor(Tensor* dst, Tensor* src){
    if(dst->device_type != src->device_type){
        printf("device_type does not match.\n");
        return NULL;
    }
    if(dst->num_dim <2){
        printf("dimention lesser than 2 dimention need no Transpose.\n");
        return NULL;
    }
    if(dst->num_dim != src->num_dim){
        printf("num_dim does not match.\n");
        return NULL;
    }
    
    for(int i=0; i < dst->num_dim - 2; i++){
        if(dst->dim[i] != src->dim[i]){
            printf("dimension %d does not match.\n", i);
            return NULL;
        }
    }
    if(dst->dim[dst->num_dim - 1] != src->dim[src->num_dim-2] || dst->dim[dst->num_dim - 2] != src->dim[src->num_dim-1]){
        printf("row and col does not match.\n");
        return NULL;
    }
    

    if(dst->isSub){
        printf("dst sub Matrix not allowed\n");
        return NULL;
    }
    
    if(dst->device_type){
        int dim_mat_dst;
        int dim_mat_src;
        int z;

        if(dst->num_dim == 2){
            dim_mat_dst = dst->sizeTensor;
            dim_mat_src = src->sizeTensor;
            z = 1;
        }else{
            dim_mat_dst = dst->stride[dst->num_dim - 3];
            dim_mat_src = src->stride[dst->num_dim - 3];
            z = dst->sizeTensor / dst->stride[dst->num_dim - 3];
        }
        hipSetDevice(dst->device_type-1);
        dim3 dimGrid((dst->dim[dst->num_dim - 1] + tile_SIZE - 1) / tile_SIZE, (dst->dim[dst->num_dim - 2] + tile_SIZE - 1) / tile_SIZE, z); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
        dim3 dimBlock(tile_SIZE,1);

        transposeCoalesced<<<dimGrid, dimBlock>>>(dst->T, src->T, dst->dim[dst->num_dim - 2], dst->dim[dst->num_dim - 1], src->stride[src->num_dim - 2], dim_mat_dst, dim_mat_src);
    }else{
        if(dst->num_dim == 1){
            for(int i=0; i < dst->sizeTensor; i++){
                dst->dim[i] = src->dim[i];
            }
        }
        else if(dst->num_dim == 2){
            for(int i=0; i < dst->dim[0]; i++){
                for(int j=0; j < dst->dim[1]; j++){
                    dst->T[i*dst->dim[1] + j] = src->T[j*src->stride[0] + i];
                }
            }
        }
        else{
            
            for(int k=0; k < dst->sizeTensor / dst->stride[dst->num_dim - 3]; k++){
                for(int i=0; i < dst->dim[dst->num_dim - 2]; i++){
                    for(int j=0; j < dst->dim[dst->num_dim - 1]; j++){
                        dst->T[k * dst->stride[dst->num_dim - 3] + i * dst->stride[dst->num_dim - 2] + j] = src->T[k * src->stride[src->num_dim - 3] + j * src->stride[src->num_dim - 2] + i];
                    }
                }
            }
        }
    }
    return dst;

}

//이거 num_dim 3이상이어야 한다.
//This has to have Z dim value of C->sizeTensor / C->stride[C->num_dim - 2]
__global__ void compTiledMM_Abig(float*A, float *B, float *C, float *bias, int *dimA, int *dimB, int *dimC, int num_dim, int little_num_dim, char bias_row){
    int matIdx_A = 0, matIdx_B = 0;
    int matIdx_C = blockDim.z * blockIdx.z; //We multiply matrixDim because We have to calculate the IdxA, IdxB.
    int cont = num_dim - little_num_dim;

 int matIdx_tmp = matIdx_C * dimC[num_dim + num_dim - 3];

for(int i=0; i < cont; i++){             //To matrix(이거 그냥 가져다 쓰면 된다. 뭐 곱할 필요 없음.)
    if(dimA[i] != 1){
        matIdx_A += (matIdx_tmp/dimC[num_dim + i]) * dimA[num_dim + i];// dim inx 값을 도출하여 각각의 stride를 곱해준다. 
    }
    matIdx_tmp %= dimC[num_dim + i];
}

for(int i=cont; i < num_dim - 2; i++){      //To matrix
    if(dimA[i] != 1){
        matIdx_A += (matIdx_tmp/dimC[num_dim + i]) * dimA[num_dim + i];// dim inx 값을 도출하여 각각의 stride를 곱해준다. 
    }
    if(dimB[i-cont] != 1){
        matIdx_B += (matIdx_tmp/dimC[num_dim + i]) * dimB[little_num_dim + i-cont];
    }
    matIdx_tmp %= dimC[num_dim + i];
}
    
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float s_a[tile_SIZE][tile_SIZE];
    __shared__ float s_b[tile_SIZE][tile_SIZE];

    float tmp = 0.0f;
    
    //output 매트릭스 stride는 그냥 dim에 맞게 해라. 그게 맞다,,,, 귀찮게 하지 말고.....

    for (int i = 0; i < (dimA[num_dim - 1] + tile_SIZE - 1) / tile_SIZE; i++) {
        //A와 B에 데이터를 넣을 때 matIdx에 차이를 두어야 한다.
        //예를 들자면 matIdx_A = blockDim.z * blockIdx.z / big_Dim_stride; 이러면 반복이 되니까.
        if (row < dimA[num_dim - 2] && (i * tile_SIZE + threadIdx.x) < dimA[num_dim - 1])
            s_a[threadIdx.y][threadIdx.x] = A[matIdx_A + row * dimA[2*num_dim - 2] + (i * tile_SIZE + threadIdx.x)];
        else 
            s_a[threadIdx.y][threadIdx.x] = 0.0f;
        

        if (col < dimB[little_num_dim-1] && (i * tile_SIZE + threadIdx.y) < dimA[num_dim-1])
            s_b[threadIdx.y][threadIdx.x] = B[matIdx_B  + (i * tile_SIZE + threadIdx.y) * dimB[2*little_num_dim - 2] + col];
        else
            s_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int j = 0; j < tile_SIZE; j++) {
            tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
        }

        __syncthreads();
    }

    // if(row==0 && col==0 && matIdx_C == 0){
    //     for(int i=0; i < tile_SIZE; i++){
    //         for(int j=0; j < tile_SIZE; j++){
    //             printf("%.02f\t", s_a[i][j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    //     for(int i=0; i < tile_SIZE; i++){
    //         for(int j=0; j < tile_SIZE; j++){
    //             printf("%.02f\t", s_b[i][j]);
    //         }
    //         printf("\n");
    //     }
    // }
    // printf("[%d %d] [%d %d]\n", row, col, dimC[num_dim - 2], dimC[num_dim-1]);
    if (row < dimC[num_dim - 2] && col < dimC[num_dim - 1]) {
        int bias_row = 0;
        if (bias)
            tmp = tmp + bias[(bias_row?row:col)];
        // printf("[%d %d %d] [%d %d] : %f\n", matIdx_C,row, col, dimC[num_dim - 2], dimC[num_dim-1], tmp);
        C[matIdx_C * dimC[2*num_dim - 3] + row * dimC[2*num_dim - 2] + col] = tmp;
    }

}

__global__ void compTiledMM_Bbig(float*A, float *B, float *C, float *bias, int *dimA, int *dimB, int *dimC, int little_num_dim, int num_dim, char bias_row){
    int matIdx_A = 0, matIdx_B = 0;
    int matIdx_C = blockDim.z * blockIdx.z; //We multiply matrixDim because We have to calculate the IdxA, IdxB.
    int cont = num_dim - little_num_dim;

 int matIdx_tmp = matIdx_C * dimC[num_dim + num_dim - 3];

for(int i=0; i < cont; i++){             //To matrix(이거 그냥 가져다 쓰면 된다. 뭐 곱할 필요 없음.)
    if(dimB[i] != 1){
        matIdx_B += (matIdx_tmp/dimC[num_dim + i]) * dimB[num_dim + i];// dim inx 값을 도출하여 각각의 stride를 곱해준다. 
    }
    matIdx_tmp %= dimC[num_dim + i];
}

for(int i=cont; i < num_dim - 2; i++){      //To matrix
    if(dimA[i-cont] != 1){
        matIdx_A += (matIdx_tmp/dimC[num_dim + i]) * dimA[little_num_dim + i-cont];// dim inx 값을 도출하여 각각의 stride를 곱해준다. 
    }
    if(dimB[i] != 1){
        matIdx_B += (matIdx_tmp/dimC[num_dim + i]) * dimB[num_dim + i];
    }
    matIdx_tmp %= dimC[num_dim + i];
}
    
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float s_a[tile_SIZE][tile_SIZE];
    __shared__ float s_b[tile_SIZE][tile_SIZE];

    float tmp = 0.0f;
    
    //output 매트릭스 stride는 그냥 dim에 맞게 해라. 그게 맞다,,,, 귀찮게 하지 말고.....

    for (int i = 0; i < (dimA[little_num_dim - 1] + tile_SIZE - 1) / tile_SIZE; i++) {
        //A와 B에 데이터를 넣을 때 matIdx에 차이를 두어야 한다.
        //예를 들자면 matIdx_A = blockDim.z * blockIdx.z / big_Dim_stride; 이러면 반복이 되니까.
        if (row < dimA[little_num_dim - 2] && (i * tile_SIZE + threadIdx.x) < dimA[little_num_dim - 1])
            s_a[threadIdx.y][threadIdx.x] = A[matIdx_A + row * dimA[2*little_num_dim - 2] + (i * tile_SIZE + threadIdx.x)];
        else 
            s_a[threadIdx.y][threadIdx.x] = 0.0f;
        

        if (col < dimB[num_dim-1] && (i * tile_SIZE + threadIdx.y) < dimA[num_dim-1])
            s_b[threadIdx.y][threadIdx.x] = B[matIdx_B  + (i * tile_SIZE + threadIdx.y) * dimB[2*num_dim - 2] + col];
        else
            s_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int j = 0; j < tile_SIZE; j++) {
            tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < dimC[num_dim - 2] && col < dimC[num_dim - 1]) {
        if (bias)
            tmp = tmp + bias[(bias_row?row:col)];
        // printf("[%d %d %d] [%d %d] : %f\n", matIdx_C,row, col, dimC[num_dim - 2], dimC[num_dim-1], tmp);
        C[matIdx_C * dimC[2*num_dim - 3] + row * dimC[2*num_dim - 2] + col] = tmp;
    }

}

__global__ void tiledMM_Half_bigA(float *A, float *B, float *C, float *bias, int* A_dim_stride, int *B_dim_stride, int C_mat_dim, int num_dim_A, int num_dim_B, char bias_row){
    
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    
    __shared__ float s_a[tile_SIZE][tile_SIZE];
    __shared__ float s_b[tile_SIZE][tile_SIZE];

    int A_matidx = blockIdx.z;
    
    float tmp = 0.0f;
    //K = A_dim_stride[A->num_dim - 1]
    //M = A_dim_stride[A->num_dim - 2]
    //N = B_dim_stride[B->num_dim - 1]
    for (int i = 0; i < (A_dim_stride[num_dim_A - 1] + tile_SIZE - 1) / tile_SIZE; i++) {
        if (row < A_dim_stride[num_dim_A - 2] && (i * tile_SIZE + threadIdx.x) < A_dim_stride[num_dim_A - 1])
        //A와 B에 데이터를 넣을 때 matIdx에 차이를 두어야 한다.
        //예를 들자면 matIdx_A = blockDim.z * blockIdx.z / big_Dim_stride; 이러면 반복이 되니까.
            s_a[threadIdx.y][threadIdx.x] = A[A_matidx * A_dim_stride[num_dim_A + num_dim_A - 3] + row * A_dim_stride[num_dim_A + num_dim_A - 2] + (i * tile_SIZE + threadIdx.x)];
        else
            s_a[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < B_dim_stride[num_dim_B - 1] && (i * tile_SIZE + threadIdx.y) < A_dim_stride[num_dim_A - 1])
            s_b[threadIdx.y][threadIdx.x] = B[(i * tile_SIZE + threadIdx.y) * B_dim_stride[num_dim_B + num_dim_B - 2] + col];
        else
            s_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();
        // if(row==0 && col== 0 && A_matidx == 1){
        // }
        for (int j = 0; j < tile_SIZE; j++) {
            tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < A_dim_stride[num_dim_A - 2] && col < B_dim_stride[num_dim_B - 1]) {
        if (bias)
            tmp = tmp + bias[(bias_row?row:col)];
        C[blockIdx.z * C_mat_dim + row * B_dim_stride[num_dim_B - 1] + col] = tmp;
        
    }

}




__global__ void tiledMM_Half_bigB(float *A, float *B, float *C, float *bias, int* A_dim_stride, int *B_dim_stride, int C_mat_dim, int num_dim_A, int num_dim_B, char bias_row){
    
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    
    __shared__ float s_a[tile_SIZE][tile_SIZE];
    __shared__ float s_b[tile_SIZE][tile_SIZE];

    int B_matidx =  blockIdx.z;
    
    float tmp = 0.0f;
    //K = A_dim_stride[A->num_dim - 1]
    //M = A_dim_stride[A->num_dim - 2]
    //N = B_dim_stride[B->num_dim - 1]
    for (int i = 0; i < (A_dim_stride[num_dim_A - 1] + tile_SIZE - 1) / tile_SIZE; i++) {
        if (row < A_dim_stride[num_dim_A - 2] && (i * tile_SIZE + threadIdx.x) < A_dim_stride[num_dim_A - 1])
        //A와 B에 데이터를 넣을 때 matIdx에 차이를 두어야 한다.
        //예를 들자면 matIdx_A = blockDim.z * blockIdx.z / big_Dim_stride; 이러면 반복이 되니까.
            s_a[threadIdx.y][threadIdx.x] = A[row * A_dim_stride[num_dim_A + num_dim_A - 2] + (i * tile_SIZE + threadIdx.x)];
        else
            s_a[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < B_dim_stride[num_dim_B - 1] && (i * tile_SIZE + threadIdx.y) < A_dim_stride[num_dim_A - 1])
            s_b[threadIdx.y][threadIdx.x] = B[B_matidx * B_dim_stride[num_dim_B + num_dim_B - 3] + (i * tile_SIZE + threadIdx.y) * B_dim_stride[num_dim_B + num_dim_B - 2] + col];
        else
            s_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int j = 0; j < tile_SIZE; j++) {
            tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < A_dim_stride[num_dim_A - 2] && col < B_dim_stride[num_dim_B - 1]) {
        if (bias)
            tmp = tmp + bias[(bias_row?row:col)];
        C[blockIdx.z * C_mat_dim + row * B_dim_stride[num_dim_B - 1] + col] = tmp;
    }

}



__global__ void tiledMM_2d(float *A, float *B, float *C, float *bias, int M, int N, int K, int A_stride, int B_stride, char bias_row) {    
    
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float s_a[tile_SIZE][tile_SIZE];
    __shared__ float s_b[tile_SIZE][tile_SIZE];

    float tmp = 0.0f;

    for (int i = 0; i < (K + tile_SIZE - 1) / tile_SIZE; i++) {
        if (row < M && (i * tile_SIZE + threadIdx.x) < K)
        //A와 B에 데이터를 넣을 때 matIdx에 차이를 두어야 한다.
        //예를 들자면 matIdx_A = blockDim.z * blockIdx.z / big_Dim_stride; 이러면 반복이 되니까.
            s_a[threadIdx.y][threadIdx.x] = A[row * A_stride + (i * tile_SIZE + threadIdx.x)];
        else
            s_a[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < N && (i * tile_SIZE + threadIdx.y) < K)
            s_b[threadIdx.y][threadIdx.x] = B[(i * tile_SIZE + threadIdx.y) * B_stride + col];
        else
            s_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        for (int j = 0; j < tile_SIZE; j++) {
            tmp += s_a[threadIdx.y][j] * s_b[j][threadIdx.x];
        }

        __syncthreads();

    }

    if (row < M && col < N) {
        if (bias)
            tmp = tmp + bias[(bias_row?row:col)];
        C[row * N + col] = tmp;
    }
}

__global__ void checkmem(float* arr, int len){
    for(int i=0; i < len; i++){
        printf("%f ", arr[i]);
    }
    printf("\n");
}

Tensor* matmul(Tensor* dC, Tensor* dA, Tensor* dB){
    if(!dC||!dB||!dA){
        printf("one of Tensor is NULL.\n");
        return NULL;
    }
    if(dC->isSub){
        printf("The result Matrix can not be a sub Matrix.\n");
        return NULL;
    }
    if(dC->device_type != dB->device_type || dB->device_type != dA->device_type){
        printf("Three matrices are in different device.");
        return NULL;
    }

    
    if(dC->num_dim <2 || dB->num_dim <2 || dA->num_dim <2){
        printf("One of three Tensor has less than 2 dimentions.(Try to use matmul().)\n");
        return NULL;
    }
    

    //2. matrix col and row has to be the same.
    if(dA->dim[dA->num_dim - 1] != dB->dim[dB->num_dim - 2]){
        printf("tensor's row and column does not match.(%d %d)\n", dA->dim[dA->num_dim - 1], dB->dim[dB->num_dim - 2]);
        return NULL;
    }

    if(dA->dim[dA->num_dim - 2] != dC->dim[dC->num_dim - 2] || dB->dim[dB->num_dim - 1] != dC->dim[dC->num_dim - 1]){
        printf("tensor's row and column does not match.\n");
        return NULL;
    }
    
    Tensor* bigTensor, *smallTensor;

    if(dA->num_dim >= dB->num_dim){
        bigTensor = dA;
        smallTensor = dB;
    }else{
        bigTensor = dB;
        smallTensor = dA;
    }


    if(bigTensor->num_dim != dC->num_dim){
        printf("dC num_dim has to have same num_dim as bigger Tensor.\n");
        return NULL;
    }

    if(bigTensor->num_dim ==2){
        hipSetDevice(bigTensor->device_type-1);
        dim3 dimGrid((dB->dim[dB->num_dim - 1] + tile_SIZE - 1) / tile_SIZE, (dA->dim[dA->num_dim - 2] + tile_SIZE - 1) / tile_SIZE, 1); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
        dim3 dimBlock(tile_SIZE, tile_SIZE);
        tiledMM_2d<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, NULL, dA->dim[0], dB->dim[1], dA->dim[1], dA->stride[0], dB->stride[0], 0);
    
    }
    else{
        //num_dim이 모두 2보다 클 때
        for(int i=0; i < bigTensor->num_dim - 2; i++){
            if(bigTensor->dim[i] != dC->dim[i] && bigTensor->dim[i] != 1){
                printf("%d dimention of Tensor(big) does not Match.\n", i);
                return NULL;
            }
        }
        
        if(smallTensor->num_dim == 2){

            hipSetDevice(dA->device_type - 1);
            dim3 dimGrid((dC->dim[dC->num_dim - 1] + tile_SIZE - 1) / tile_SIZE, (dC->dim[dC->num_dim - 2] + tile_SIZE - 1) / tile_SIZE, dC->sizeTensor / dC->stride[bigTensor->num_dim - 3]); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
            dim3 dimBlock(tile_SIZE, tile_SIZE);
            if(dA == bigTensor)
                tiledMM_Half_bigA<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, NULL, dA->d_dim_stride, dB->d_dim_stride, dC->stride[dC->num_dim - 3], dA->num_dim, dB->num_dim, 0);
            else
                tiledMM_Half_bigB<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, NULL, dA->d_dim_stride, dB->d_dim_stride, dC->stride[dC->num_dim - 3], dA->num_dim, dB->num_dim, 0);

        }else{

            int cont = bigTensor->num_dim - smallTensor->num_dim;

            //1. The tensor has to match if one of them is not 1.
            for(int i=0; i < smallTensor->num_dim - 2; i++){
                if(smallTensor->dim[i] != dC->dim[i+cont] && smallTensor->dim[i] != 1){
                    printf("%d dimention of Tensor(small) does not Match.\n", i);
                    return NULL;
                }
            }

            hipSetDevice(dA->device_type - 1);
            dim3 dimGrid((dC->dim[dC->num_dim - 1] + tile_SIZE - 1) / tile_SIZE, (dC->dim[dC->num_dim - 2] + tile_SIZE - 1) / tile_SIZE, dC->sizeTensor / dC->stride[dC->num_dim - 3]); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
            dim3 dimBlock(tile_SIZE, tile_SIZE);
            if(dA == bigTensor)
                compTiledMM_Abig<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, NULL, dA->d_dim_stride, dB->d_dim_stride, dC->d_dim_stride, dA->num_dim, dB->num_dim, 0);
            else
                compTiledMM_Bbig<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, NULL, dA->d_dim_stride, dB->d_dim_stride, dC->d_dim_stride, dA->num_dim, dB->num_dim, 0);
            
        }
        
    }
    
    return dC;
    
    //if [5, 1, 3, 4, 5] x [4, 1, 5, 4]
    

}


Tensor* matmul_bias(Tensor* dC, Tensor* dA, Tensor* dB, Tensor* dbias, char row_bias){
    if(!dC||!dB||!dA||!dbias){
        printf("one of Tensor is NULL.\n");
        return NULL;
    }
    if(dC->isSub){
        printf("The result Matrix can not be a sub Matrix.\n");
        return NULL;
    }
    if(dC->device_type != dB->device_type || dB->device_type != dA->device_type || dbias->device_type != dA->device_type){
        printf("Four matrices are in different device.");
        return NULL;
    }

    
    if(dC->num_dim <2 || dB->num_dim <2 || dA->num_dim <2){
        printf("One of three Tensor has less than 2 dimentions.(Try to use matmul().)\n");
        return NULL;
    }
    
    if(dbias->num_dim != 1){
        if(dbias->num_dim != 2 || dbias->dim[1] != dC->dim[dC->num_dim - 1]){
            printf("bias not an appropriate dimention.\n");
            return NULL;
        }
    }

    if(dbias->dim[dbias->num_dim - 1] != dB->dim[dB->num_dim - 1]){
        printf("Bias size does not fit to column(dB's last dim).\n");
        return NULL;
    }

    //2. matrix col and row has to be the same.
    if(dA->dim[dA->num_dim - 1] != dB->dim[dB->num_dim - 2]){
        printf("tensor's row and column does not match.(%d %d)\n", dA->dim[dA->num_dim - 1], dB->dim[dB->num_dim - 2]);
        return NULL;
    }

    if(dA->dim[dA->num_dim - 2] != dC->dim[dC->num_dim - 2] || dB->dim[dB->num_dim - 1] != dC->dim[dC->num_dim - 1]){
        printf("tensor's row and column does not match.\n");
        return NULL;
    }
    
    Tensor* bigTensor, *smallTensor;

    if(dA->num_dim >= dB->num_dim){
        bigTensor = dA;
        smallTensor = dB;
    }else{
        bigTensor = dB;
        smallTensor = dA;
    }


    if(bigTensor->num_dim != dC->num_dim){
        printf("dC num_dim has to have same num_dim as bigger Tensor.\n");
        return NULL;
    }

    if(bigTensor->num_dim ==2){
        hipSetDevice(bigTensor->device_type-1);
        dim3 dimGrid((dB->dim[dB->num_dim - 1] + tile_SIZE - 1) / tile_SIZE, (dA->dim[dA->num_dim - 2] + tile_SIZE - 1) / tile_SIZE, 1); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
        dim3 dimBlock(tile_SIZE, tile_SIZE);
        tiledMM_2d<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, dbias->T, dA->dim[0], dB->dim[1], dA->dim[1], dA->stride[0], dB->stride[0], row_bias);
    
    }
    else{
        //num_dim이 모두 2보다 클 때
        for(int i=0; i < bigTensor->num_dim - 2; i++){
            if(bigTensor->dim[i] != dC->dim[i] && bigTensor->dim[i] != 1){
                printf("%d dimention of Tensor(big) does not Match.\n", i);
                return NULL;
            }
        }
        
        if(smallTensor->num_dim == 2){

            hipSetDevice(dA->device_type - 1);
            dim3 dimGrid((dC->dim[dC->num_dim - 1] + tile_SIZE - 1) / tile_SIZE, (dC->dim[dC->num_dim - 2] + tile_SIZE - 1) / tile_SIZE, dC->sizeTensor / dC->stride[bigTensor->num_dim - 3]); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
            dim3 dimBlock(tile_SIZE, tile_SIZE);
            if(dA == bigTensor)
                tiledMM_Half_bigA<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, dbias->T, dA->d_dim_stride, dB->d_dim_stride, dC->stride[dC->num_dim - 3], dA->num_dim, dB->num_dim, row_bias);
            else
                tiledMM_Half_bigB<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, dbias->T, dA->d_dim_stride, dB->d_dim_stride, dC->stride[dC->num_dim - 3], dA->num_dim, dB->num_dim, row_bias);

        }else{

            int cont = bigTensor->num_dim - smallTensor->num_dim;

            //1. The tensor has to match if one of them is not 1.
            for(int i=0; i < smallTensor->num_dim - 2; i++){
                if(smallTensor->dim[i] != dC->dim[i+cont] && smallTensor->dim[i] != 1){
                    printf("%d dimention of Tensor(small) does not Match.\n", i);
                    return NULL;
                }
            }

            hipSetDevice(dA->device_type - 1);
            dim3 dimGrid((dC->dim[dC->num_dim - 1] + tile_SIZE - 1) / tile_SIZE, (dC->dim[dC->num_dim - 2] + tile_SIZE - 1) / tile_SIZE, dC->sizeTensor / dC->stride[dC->num_dim - 3]); //dim은 4x3x32x32를 matmul하는 경우 12가 들어가게 된다.
            dim3 dimBlock(tile_SIZE, tile_SIZE);
            if(dA == bigTensor)
                compTiledMM_Abig<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, dbias->T, dA->d_dim_stride, dB->d_dim_stride, dC->d_dim_stride, dA->num_dim, dB->num_dim, row_bias);
            else
                compTiledMM_Bbig<<<dimGrid, dimBlock>>>(dA->T, dB->T, dC->T, dbias->T, dA->d_dim_stride, dB->d_dim_stride, dC->d_dim_stride, dA->num_dim, dB->num_dim, row_bias);
            
        }
        
    }
    return dC;
    
    //if [5, 1, 3, 4, 5] x [4, 1, 5, 4]
    

}










__global__ void ReLU(float* T, int sizeTensor){
    int inx = blockIdx.x * blockDim.x + threadIdx.x;
    if(inx < sizeTensor)
        T[inx] = (T[inx] >=0) ? T[inx] : 0;
}


Tensor* ReLU_inline(Tensor* ten){
    if(!ten){
        printf("No Tensor.\n");
        return NULL;
    }
    if(ten->device_type){
        hipSetDevice(ten->device_type - 1);
        ReLU<<<(ten->sizeTensor+tile_SIZE-1)/tile_SIZE, tile_SIZE*tile_SIZE>>>(ten->T, ten->sizeTensor);
    }else{
        for(int i=0; i < ten->sizeTensor; i++){
            ten->T[i] = (ten->T[i] >=0) ? ten->T[i] : 0;
        }
        
    }
    return ten;
}

__global__ void gelu_(float* in, float* out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < len) {  
        // Do the actual computation
        out[i] = 0.5 * in[i] * (1.0 + tanh(0.79788456 * (in[i] + 0.044715 * in[i] * in[i] * in[i])));
    }
}


// __global__ void sub_(float* dC, float* dA, float* dB, int len){
//     int inx = blockDim.x * blockIdx.x + threadIdx.x;
// }

// Tensor* subtract_Tensor(Tensor* dC, Tensor* dB, Tensor* dA){
//     if(!dC || !dA || !dB){
//         printf("no Tensor\n");
//         return NULL;
//     }
//     if(dC->device_type != dB->device_type && dB->device_type != dA->device_type){
//         printf("Device type does not match\n");
//         return NULL;
//     }
    
//     if(dC->num_dim != dB->num_dim || dC->num_dim != dA->num_dim){
//         printf("Number of dim does not match.\n");
//         return NULL;
//     }
//     for(int i=0; i < dC->num_dim; i++){
//         if(dC->dim[i] != dB[i] || dC->dim[i] != dA->dim[i]){
//             printf("dimention does not match.\n");
//             return NULL;
//         }
//     }

//     if(dC->device_type){

//     }else{

//     }
//     return dC;
// }